#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void kernel() {

    int tid = threadIdx.x;

    if (tid < 8) {
        printf("inside the kernel\n");
    }
    else {
        printf("outside the kernel\n");
    }

}

int cuda(int a, int b) {
    kernel<<<1, 10>>>();
    hipDeviceSynchronize();

    return 0;
}


