#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>


template <typename T>
__global__ void matrixMultiplicationKernel(T *A, T *B, T *C, int N) {
    
	int ROW = blockIdx.y * blockDim.y + threadIdx.y;
	int COL = blockIdx.x * blockDim.x + threadIdx.x;

	T tmpSum = 0;

	if (ROW < N && COL < N) {
	    for (int i = 0; i < N; i++) {
		    tmpSum += A[ROW * N + i] * B[i * N + COL];
		}
	}
	C[ROW * N + COL] = tmpSum;
}


template <typename T>
void matrixMultiplication(float *A, float *B, float *C, int M, int K, int N) {

	// declare the number of blocks per grid and the number of threads per block
	// use 1 to 512 threads per block
	dim3 threadsPerBlock(N, N);
	dim3 blocksPerGrid(1, 1);
	if (N*N > 512) {
	    threadsPerBlock.x = 512;
		threadsPerBlock.y = 512;
		blocksPerGrid.x = ceil(double(N)/double(threadsPerBlock.x));
		blocksPerGrid.y = ceil(double(N)/double(threadsPerBlock.y));
	}

	matrixMultiplicationKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, N);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
	    std::stringstream strstr;
		strstr << "matrixMultiplicationKernel launch failed" << std::endl;
		strstr << "dimBlock: " << blocksPerGrid.x << ", " << blocksPerGrid.y << std::endl;
		strstr << "dimGrid: " << threadsPerBlock.x << ", " << threadsPerBlock.y << std::endl;
		strstr << hipGetErrorString(error);
		throw strstr.str();
	}
}

template <typename T>
void map_matrix(pybind11::array_t<T> A, pybind11::array_t<T> B, pybind11::array_t<T> C, int M, int K, int N) {
	pybind11::buffer_info AA = A.request();
	pybind11::buffer_info BB = B.request();
	pybind11::buffer_info CC = C.request();

	if (AA.ndim != 1) {
	    std::stringstream strstr;
		strstr << "AA.ndim != 1" << std::endl;
		strstr << "AA.ndim: " << AA.ndim << std::endl;
		throw std::runtime_error(strstr.str());
	}

	if (BB.ndim != 1) {
	    std::stringstream strstr;
		strstr << "BB.ndim != 1" << std::endl;
		strstr << "BB.ndim: " << BB.ndim << std::endl;
		throw std::runtime_error(strstr.str());
	}

	if (CC.ndim != 1) {
	    std::stringstream strstr;
		strstr << "CC.ndim != 1" << std::endl;
		strstr << "CC.ndim: " << CC.ndim << std::endl;
		throw std::runtime_error(strstr.str());
	}

	int size_A = AA.shape[0];
	if (M*N != size_A) {
	    std::stringstream strstr;
		strstr << "Matrix A size != M * N" << std::endl;
		strstr << "Please confirm the shape" << std::endl;
		throw std::runtime_error(strstr.str());
	}

	int size_B = BB.shape[0];
	if (N*K != size_B) {
        std::stringstream strstr;
		strstr << "Matrix B size != N * K" << std::endl;
		strstr << "Please confirm the shape" << std::endl;
		throw std::runtime_error(strstr.str());
	}

	int size_C = CC.shape[0];
	if (M*N != size_C) {
	    std::stringstream strstr;
		strstr << "Matrix C size != M * K" << std::endl;
		strstr << "Please confirm the shape" << std::endl;
		throw std::runtime_error(strstr.str());
	}

	int size_bytes_A = size_A * sizeof(T);
	int size_bytes_B = size_B * sizeof(T);
	int size_bytes_C = size_C * sizeof(T);

	T *AGpu_ptr;
	T *BGpu_ptr;
	T *CGpu_ptr;

	hipError_t error = hipMalloc((void**)&AGpu_ptr, size_bytes_A);
	if (error != hipSuccess) {
	    throw std::runtime_error(hipGetErrorString(error));
	}

	error = hipMalloc((void**)&BGpu_ptr, size_bytes_B);
	if (error != hipSuccess) {
	    throw std::runtime_error(hipGetErrorString(error));
	}

	error = hipMalloc((void**)&CGpu_ptr, size_bytes_C);
	if (error != hipSuccess) {
	    throw std::runtime_error(hipGetErrorString(error));
	}

	T* ptr_A = reinterpret_cast<T*>(AA.ptr);
	T* ptr_B = reinterpret_cast<T*>(BB.ptr);
	T* ptr_C = reinterpret_cast<T*>(CC.ptr);

	error = hipMemcpy(AGpu_ptr, ptr_A, size_bytes_A, hipMemcpyHostToDevice);

	if (error != hipSuccess) {
	    throw std::runtime_error(hipGetErrorString(error));
	}

	error = hipMemcpy(BGpu_ptr, ptr_B, size_bytes_B, hipMemcpyHostToDevice);

	if (error != hipSuccess) {
	    throw std::runtime_error(hipGetErrorString(error));
	}

	matrixMultiplication<T>(AGpu_ptr, BGpu_ptr, CGpu_ptr, M, K, N);

	error = hipMemcpy(ptr_C, CGpu_ptr, size_bytes_C, hipMemcpyDeviceToHost);

	if (error != hipSuccess) {
	    throw std::runtime_error(hipGetErrorString(error));
	}
}

int add(int i, int j) {
    return i + j;
}

PYBIND11_MODULE(matrix_multi_library, m) {
    m.doc() = "matrix multi using GPU";
	m.def("add", &add, "multi two matrix");
	m.def("matrix_multi", map_matrix<float>, "multi two matrix");
}
