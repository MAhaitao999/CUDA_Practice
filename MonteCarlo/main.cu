
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctime>

#define CUDA_KERNEL_LOOP(i, n) \
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
		 i < (n); \
		 i += blockDim.x * gridDim.x)

__global__ void distance(float *xSquare, float *ySquare, int *result, int testNum) {
    // int tid = threadIdx.x;
	// int bid = blockIdx.x;
	// int idx = tid + blockDim.x * bid;
	// printf("idx is: %d\n", idx);
	CUDA_KERNEL_LOOP(index, testNum) {
	    // if (index < num) 
		{
		    float temp = (xSquare[index] - 1) * (xSquare[index] - 1) + (ySquare[index] - 1) * (ySquare[index] - 1);
			if (temp < 1) {
			    result[index] = 1;
			}
			else {
			    result[index] = 0;
				// printf("**********************************************************");
			}
		}
	}
}

__global__ void sum_array(int *a, int *b, int testNum) {
    
	int tid = threadIdx.x;
	b[0] = 0;
	__shared__ float sData[512];

	for (int count = 0; count < ceilf(testNum / 512); count++) {
	    if (tid + count * 512 < testNum) {
		    sData[tid] = a[tid + count * 512];
			__syncthreads();
		}

		for (int i = 512 / 2; i > 0; i /= 2) {
		    if (tid < i) {
			    sData[tid] = sData[tid] + sData[tid + i];
			}
			__syncthreads();
		}

		if (tid == 0) {
		    b[0] += sData[0];
		}
	}
}

int main(int argc, char* argv[]) {

	int testNum = 100000000;
	srand((int)time(0));

	float *xSquare = new float[testNum];
	float *ySquare = new float[testNum];

	for (int i = 0; i < testNum; i++) {
	    xSquare[i] = rand() % 10000 * 1.0 / 10000;
		ySquare[i] = rand() % 10000 * 1.0 / 10000;
	}

	float *xSquareGpu;
	hipMalloc((void**)&xSquareGpu, testNum * sizeof(float));
	hipMemcpy(xSquareGpu, xSquare, testNum * sizeof(float), hipMemcpyHostToDevice);

	float *ySquareGpu;
	hipMalloc((void**)&ySquareGpu, testNum * sizeof(float));
	hipMemcpy(ySquareGpu, ySquare, testNum * sizeof(float), hipMemcpyHostToDevice);

	int threadNum = 1024;
	int blockNum = 512;
	int *resultGpu;
	hipMalloc((void**)&resultGpu, testNum * sizeof(int));
	distance<<<blockNum, threadNum>>>(xSquareGpu, ySquareGpu, resultGpu, testNum);

	int *result = new int[testNum];
	hipMemcpy(result, resultGpu, testNum * sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < 10; i++) {
	    printf("result[%d] is %d\n", i, result[i]);
	}

	int *bGpu;
	hipMalloc((void**)&bGpu, 1 * sizeof(int));

	sum_array<<<1, 512>>>(resultGpu, bGpu, testNum);

	int b[1];
	hipMemcpy(b, bGpu, 1 * sizeof(int), hipMemcpyDeviceToHost);

	printf("the b[0] is: %d\n", b[0]);
	printf("the appropriate pi is: %2.6f\n", float(b[0])/testNum*4);
    
	return 0;
}
