
#include <hip/hip_runtime.h>
#include <stdio.h>

#define LENGTH 16
#define THREADNUM 4
#define BLOCKNUM 2

static void HandleError(hipError_t err,
		       const char *file,
		       int line) {
                           if (err != hipSuccess) {
			       printf("%s in %s at line %d\n",
			       hipGetErrorString(err),
			       file, line);
			       exit(EXIT_FAILURE);
			   }
                       }

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

int getThreadNum() {
    hipDeviceProp_t prop;
    int count;

    HANDLE_ERROR(hipGetDeviceCount(&count));
    printf("gpu num %d\n", count);
    HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
    printf("max thread num: %d\n", prop.maxThreadsPerBlock);
    printf("max grid dimensions: (%d, %d, %d)\n",
        prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);

    return prop.maxThreadsPerBlock;
}

__global__ void dot_product(float *a, float *b, float *r) {
    
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int total_thread_nnum = THREADNUM * BLOCKNUM;

	__shared__ float sData[THREADNUM];
	sData[tid] = 0;
	int global_id = tid + bid * blockDim.x;

	while (global_id < LENGTH) {
	    sData[tid] += a[global_id] * b[global_id];
		global_id += total_thread_nnum;
	}
	__syncthreads();
	
	for (int i = THREADNUM / 2; i > 0; i /= 2) {
	
		if (tid < i) {
		    sData[tid] = sData[tid] + sData[tid + i];
		}
		__syncthreads();
	}

	if (tid == 0) {
	    r[bid] = sData[0];
	}

}

int main(int argc, char* argv[]) {
    
	float a[LENGTH];
	float b[LENGTH];
	for (int i = 0; i < LENGTH; i++) {
	    a[i] = i * (i + 1);
		b[i] = i * (i - 2);
	}

	float *aGpu;
	hipMalloc((void**)&aGpu, LENGTH * sizeof(float));
	hipMemcpy(aGpu, a, LENGTH * sizeof(float), hipMemcpyHostToDevice);

	float *bGpu;
	hipMalloc((void**)&bGpu, LENGTH * sizeof(float));
	hipMemcpy(bGpu, b, LENGTH * sizeof(float), hipMemcpyHostToDevice);

	float *rGpu;
	hipMalloc((void**)&rGpu, BLOCKNUM * sizeof(float));

	dot_product<<<BLOCKNUM, THREADNUM>>>(aGpu, bGpu, rGpu);

	float r[BLOCKNUM];
	hipMemcpy(r, rGpu, BLOCKNUM * sizeof(float), hipMemcpyDeviceToHost);

	float result = 0.0;
	for (int i = 0; i < BLOCKNUM; i++) {
	    printf("r[%d]: %f\n", i, r[i]);
		result += r[i];
	}

	printf("result is: %f\n", result);

	return 0;

}
