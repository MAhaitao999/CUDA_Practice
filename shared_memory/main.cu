
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c, int num) {
    int i = threadIdx.x;
    // int nums = 10;
    // __shared__ int sh[nums];
    // __shared__ int* sh = new int[nums];
    // 动态分配共享内存
    extern __shared__ int sh[];
    int *x = (int *)sh;
    
    if (i < num) {
	x[i] = a[i] + b[i];
        c[i] = x[i];
    }
}

int main(int argc, char* argv[]) {

    int num = 10;
    int a[num], b[num], c[num];
    int *a_gpu, *b_gpu, *c_gpu;

    for (int i = 0; i < num; i++) {
        a[i] = i;
	b[i] = i*i;
    }

    hipMalloc((void**)&a_gpu, num*sizeof(int));
    hipMalloc((void**)&b_gpu, num*sizeof(int));
    hipMalloc((void**)&c_gpu, num*sizeof(int));

    // copy data
    hipMemcpy(a_gpu, a, num*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b, num*sizeof(int), hipMemcpyHostToDevice);

    // do
    add<<<1, num, num>>>(a_gpu, b_gpu, c_gpu, num); // <<<>>>第三个参数是共享内存大小

    // get data
    hipMemcpy(c, c_gpu, num*sizeof(int), hipMemcpyDeviceToHost);

    // visualization
    for (int i=0; i < num; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    return 0;
}
