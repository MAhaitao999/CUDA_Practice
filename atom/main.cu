
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void sum_test(float *a, float *b) {
    
    int tid = threadIdx.x;

    b[0] = 0;
    __syncthreads();
    // b[0] = a[tid] + 2;
    // printf("a[%d] is %2.2f\n", tid, a[tid]);
    // printf("the thread id is %d\n", tid);
    // printf("b[0] is: %2.1f\n", b[0]);
    // atomicAdd(&a[tid], 1);
    atomicAdd(&a[tid], 1);
    b[0] += a[tid];
}


__global__ void hist_compute(int *a, int *hist) {
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int idx = tid + bid * blockDim.x;

    // printf("a[%d] is %d\n", idx, a[idx]);

    // hist[a[idx]] += 1;
    atomicAdd(&hist[a[idx]], 1);
}

int main(int argc, char* argv[]) {

    int pixel_num = 5120;
    int a[pixel_num];
    int length = 10;
    
    for (int i = 0; i < pixel_num; i++) {
        a[i] = i * (i + 1) % length;
	// printf("a[%d]=%d\n", i, a[i]);
    }

    int *hist = new int[length]();
    
    for (int i = 0; i < pixel_num; i++) {
        hist[a[i]] += 1;
    }

    for (int i = 0; i < length; i++) {
        printf("hist[%d]=%d\n", i, hist[i]);
    }

    int *aGpu, *histGpu;
    int hist2[length];
    hipMalloc((void**)&aGpu, pixel_num * sizeof(int));
    hipMalloc((void**)&histGpu, length * sizeof(int));
    hipMemcpy(aGpu, a, pixel_num * sizeof(int), hipMemcpyHostToDevice);

    hist_compute<<<pixel_num / 512, 512>>>(aGpu, histGpu);

    hipMemcpy(hist2, histGpu, length * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < length; i++) {
        printf("hist[%d]=%d\n", i, hist2[i]);
    }

    return 0;
}
