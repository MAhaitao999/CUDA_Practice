
#include <hip/hip_runtime.h>
#include <stdio.h>

static void HandleError(hipError_t err,
		       const char *file,
		       int line) {
                           if (err != hipSuccess) {
			       printf("%s in %s at line %d\n",
			       hipGetErrorString(err),
			       file, line);
			       exit(EXIT_FAILURE);
			   }
                       }

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

int getThreadNum() {
    hipDeviceProp_t prop;
    int count;

    HANDLE_ERROR(hipGetDeviceCount(&count));
    printf("gpu num %d\n", count);
    HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
    printf("max thread num: %d\n", prop.maxThreadsPerBlock);
    printf("max grid dimensions: (%d, %d, %d)\n",
        prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);

    return prop.maxThreadsPerBlock;
}

__global__ void sum(float *a, float *b, int num_p2) {
    int tid = threadIdx.x;

    __shared__ float sData[1024];
    sData[tid] = a[tid];
    __syncthreads();

    /*
    if (tid < 8) {
        sData[tid] = sData[tid] + sData[tid + 8];
    }
    __syncthreads();

    if (tid < 4) {
        sData[tid] = sData[tid] + sData[tid + 4];
    }
    __syncthreads();

    if (tid < 2) {
        sData[tid] = sData[tid] + sData[tid + 2];
    }
    __syncthreads();

    if (tid < 1) {
        sData[tid] = sData[tid] + sData[tid + 1];
    }
    __syncthreads();

    b[0] = sData[0];
    */
    for (int i = num_p2 / 2; i > 0; i /= 2) {
        if (tid < i) {
	    sData[tid] = sData[tid] + sData[tid + i];
	}
	__syncthreads();
    }

    *b = sData[0];     
}

inline int next_p2(int a) {
    int rval = 1;
    while (rval < a) {
        rval <<= 1;
    }
    return rval;
}

int main(int argc, char* argv[]) {
    
    int num = 16;
    int num_p2 = next_p2(num);
    printf("%d's p2 is %d\n", num, num_p2);
    float a[num];
    float a_tmp[num_p2];

    for (int i = 0; i < num; i++) {
        a[i] = i * (i + 1);
    }

    for (int i = 0; i < num_p2; i++) {
        if (i < num) {
	    a_tmp[i] = a[i];
	}
	else {
	    a_tmp[i] = 0.0;
	}
    }

    float *aGpu;
    hipMalloc((void**)&aGpu, num_p2 * sizeof(float));
    hipMemcpy(aGpu, a_tmp, num_p2 * sizeof(float), hipMemcpyHostToDevice);

    float *bGpu;
    hipMalloc((void**)&bGpu, 1 * sizeof(float));
    sum<<<1, 1024>>>(aGpu, bGpu, num_p2);

    float b[1];
    hipMemcpy(b, bGpu, 1 * sizeof(float), hipMemcpyDeviceToHost);

    printf("the result is: %2.0f\n", b[0]);

    return 0;

}
